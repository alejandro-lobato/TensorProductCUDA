
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define N 3
#define M 3

//Data: a and b inputs, output: c.
int matrixA[N][M], matrixB[N][M], matrixC[N*N][M*M];

//Runs in GPU cores.
__global__ void tensorProduct(int n, int m, int *matrixA, int *matrixB, int *result)
{
  int i = threadIdx.x;
  int j = threadIdx.y;  

  //Perform tensor product.
  for (int k = 0; k < n; k++){
    for (int l = 0; l < m; l++){
      result[(j * m + l) + (i * n + k) * m * m] = matrixA[i*m+j] * matrixB[k*m+l];
    }
  }
}

void tensorProductDevice(int n, int m, int *a, int *b, int *c){

  //Parameters.
  int *aD, *bD, *cD;
  int matrixSize = n * m * sizeof(int);
  int resultSize = n * n * m * m * sizeof(int);

  dim3 blocks(1,1);
  dim3 threads(n,m);

  //1. Assign memory
  hipMalloc(&aD, matrixSize);
  hipMalloc(&bD, matrixSize);
  hipMalloc(&cD, resultSize);

  //2. Set device 0, copy the information to device.
  hipSetDevice(0);
  hipMemcpy(aD, a, matrixSize, hipMemcpyHostToDevice);
  hipMemcpy(bD, b, matrixSize, hipMemcpyHostToDevice);

  //Execute kernel.
  tensorProduct<<<blocks, threads>>>(n, m, aD, bD, cD);
  
  //Copy data from device back to host.
  hipMemcpy(c, cD, resultSize, hipMemcpyDeviceToHost);
  
  //Free memory.
  hipFree(aD);
  hipFree(bD);
  hipFree(cD);

}

int main(){

  srand(time(NULL));

  //Fill the matrices.
  for(int i = 0; i < N; i++){
    for (int j = 0; j < M; j++){
      matrixA[i][j] = 1 + rand() % 5;
      matrixB[i][j] = 1 + rand() % 5;
    }
  }

  //Call to perform tensor operation.
  tensorProductDevice(N, M, (int *) matrixA, (int *) matrixB, (int *) matrixC);
  
  
  printf("Elements of A:\n");
  for (int i = 0; i < N; i++){
    for (int j = 0; j < M; j++){
      printf("%d\t", matrixA[i][j]);
    }
    printf("\n");
  }
  printf("\n");

  printf("Elements of B:\n");
  for (int i = 0; i < N; i++){
    for (int j = 0; j < M; j++){
      printf("%d\t", matrixB[i][j]);
    }
    printf("\n");
  }
  printf("\n");

  printf("Result:\n");
  for (int i = 0; i < N * N; i++){
    for (int j = 0; j < M * M; j++){
      printf("%d\t", matrixC[i][j]);
    }
    printf("\n");
  }
  printf("\n");
}